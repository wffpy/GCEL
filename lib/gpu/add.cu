#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "gpu/gpu.h"
namespace gpu_kernel{
// CUDA kernel function to add elements of two arrays
__global__ void add(float *a, float *b, float *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}
}   // namespace gpu_kernel


namespace gpu {
int add(float *lhs, float *rhs, float *ret, int length) {
    int size = length * sizeof(float);
    // Device arrays
    float *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy data from host to device
    hipMemcpy(d_a, lhs, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, rhs, size, hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (length + blockSize - 1) / blockSize;
    gpu_kernel::add<<<numBlocks, blockSize>>>(d_a, d_b, d_c, length);

    // Copy result from device to host
    hipMemcpy(ret, d_c, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}

}   // namespace gpu