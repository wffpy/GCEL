#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "gpu/gpu.h"
namespace gpu_kernel {
// CUDA kernel function to transpose a matrix
template <typename T>
__global__ void transpose_impl_(const T* input, T* ret, int64_t row,
                                int64_t col, int64_t tile_dim,
                                int64_t block_size) {
    int x = blockIdx.x * tile_dim + threadIdx.x;
    int y = blockIdx.y * tile_dim + threadIdx.y;
    int width = gridDim.x * tile_dim;

    for (int index = 0; index < tile_dim; index += block_size) {
        ret[x * width + y + index] = input[(y + index) * width + x];
    }
}
}  // namespace gpu_kernel

namespace gpu {
template <typename T>
utils::GCELResult transpose(const T* input, T* ret, int64_t row, int64_t col) {
    int tile_dim = 32;
    int block_size = 8;
    dim3 dimGrid(row / tile_dim, col / tile_dim, 1);
    dim3 dimBlock(tile_dim, block_size, 1);

    gpu_kernel::transpose_impl_<<<dimGrid, dimBlock>>>(input, ret, row, col,
                                                       tile_dim, block_size);

    return utils::GCELResult::SUCCESS;
}

#define INSTANTIATE_TRANSPOSE(T) \
    template utils::GCELResult transpose<T>(const T*, T*, int64_t, int64_t);

INSTANTIATE_TRANSPOSE(char);
INSTANTIATE_TRANSPOSE(int32_t);
INSTANTIATE_TRANSPOSE(int64_t);
INSTANTIATE_TRANSPOSE(float);
INSTANTIATE_TRANSPOSE(double);
}  // namespace gpu